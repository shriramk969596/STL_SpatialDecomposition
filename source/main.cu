#include "hip/hip_runtime.h"
//Triangle AABB intersection detection based on Separating axis theoram.  
/* Reference: https://gamedev.stackexchange.com/questions/44500/how-many-and-which-axes-to-use-for-3d-obb-collision-with-sat
              https://gamedev.stackexchange.com/questions/118707/zero-vector-test-in-projection-intersection-test-in-sat/118718#118718
              https://bit.ly/2C7bjmM
              http://programmerart.weebly.com/separating-axis-theorem.html
              https://gdbooks.gitbooks.io/3dcollisions/content/Chapter2/static_aabb_plane.html
              https://gdbooks.gitbooks.io/3dcollisions/content/Chapter4/aabb-triangle.html
              https://fileadmin.cs.lth.se/cs/Personal/Tomas_Akenine-Moller/pubs/tribox.pdf
              http://www.dyn4j.org/2010/01/sat/#sat-proj
              http://fileadmin.cs.lth.se/cs/Personal/Tomas_Akenine-Moller/code/tribox3.txt
*/
// prefix: coord = coordinates, vec = vector  

#include <stdlib.h>
#include <vector>
#include <time.h>
#include <sstream>
#include <stdio.h>
#include "utils.h"
#include "stl_utils.h"
#include "grid_utils.h"
#include "stl_reader.h"
#include "dem_core.h"
#include "ParticleGenerator.h"
#include "z_hash.h"
#include <bitset>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_vector_types.h"
#include "triangle4.h"
#include "params.h"
#include "triangle4_utils.cuh"



using namespace std;



int main() {  

    clock_t tStart_WithGrid = clock();


    Point gridOrigin = Point(GridParam::grid_origin_x, GridParam::grid_origin_y, GridParam::grid_origin_z);
    Grid G1 = Grid(GridParam::delX, GridParam::delY, GridParam::delZ, GridParam::nx, GridParam::ny, GridParam::nz, gridOrigin);
//    G1.GenerateCellMinMaxID();
//    G1.GenerateVTK();
    //Initiating Grid Data - END

    //Assigning minimum tria size tolerance (TODO : used for filtering out triangles which need not be checked for SAT)

    cout << "READING STL MESH - START" << endl;
    stl_reader::StlMesh <float, unsigned int> mesh ("./../../../INPUTS/ball.stl");

    //Initialize Mesh Data
    TriangleMesh mesh_Tria;

    // Reading Data from .stl mesh file into TriangleMesh class
    // Vertices of the triangle (Anti-Clockwise v0,v1,v2)
    vector<float> min;
    vector<float> max;


    vector<Triangle4> trias;
    unsigned int n_trias = 0; //NUMBER OF TRIANGLES IN STL MESH
    
    //TODO: FIND TRIANGLE MESH BOUNDING BOX TO GET GRID MIN MAX
    for(size_t itri = 0; itri < mesh.num_tris(); ++itri) {

        const float* v0 = mesh.tri_corner_coords(itri, 0);
        const float* v1 = mesh.tri_corner_coords(itri, 1);
        const float* v2 = mesh.tri_corner_coords(itri, 2);
        const float* n = mesh.tri_normal(itri);

        Triangle4 tria4;

        tria4.v0 = make_float3(v0[0],v0[1], v0[2]);
        tria4.v1 = make_float3(v1[0],v1[1], v1[2]);
        tria4.v2 = make_float3(v2[0],v2[1], v2[2]);
        tria4.n = make_float3(n[0], n[1], n[2]);

        trias.push_back(tria4);

        //used only for testing portion of the code - from line 111
        Triangle tria = Triangle(Point(v0[0],v0[1],v0[2]), Point(v1[0],v1[1],v1[2]), Point(v2[0],v2[1],v2[2]),itri);
        mesh_Tria.Append(tria, itri);
        n_trias = n_trias + 1;

    }

//This portion of the code is a minimal implementation with linear cell hashing and serves as a primer of GPU implementation.///////////////////////////////
//The outcome is a vector of vectors where each subvector reprecents a cell in the cartesian grid and contains tria ID's belonging to the cell.
    vector<vector<int>> cellToTriaMap(GridParam::nx * GridParam::ny * GridParam::nz);
//    cellToTriaMap.resize(GridParam::nx * GridParam::ny * GridParam::nz);

    cout << "Building cellToTriaMap on CPU ---- START" << endl;
    int triaId = 0;
    for(Triangle4 &tria: trias) {
        TriaLocalGridIntersection(tria, triaId, cellToTriaMap);
        triaId = triaId + 1;
    }
    cout << "Building cellToTriaMap on CPU ---- END" << endl;

    //call GenerateMeshSpatialDecompositionData(.....) here to generate GPU relevant data

//This portion of the code is a minimal implementation nd and serves as a primer of GPU implementation.///////////////////////////////////////////////////


    //The piece of code above is a cleaner implementation.
    // Initial CPU implementation. It is used only for testing purposes and has some limitations.
    //The triangle size is limited such that it should not exceed beyond the 27 cell neighbourhood.
    //Initialize variables for SAT
    vector<double> vec_h;                                                          //absolute half length vector of AABB
    Vector vec_BoxCentre;                                                           //centroid of AABB
    Bbox tria_BV;
    Bbox Bbox_parentCell;                                                           //bounding volume of triangle as Bbox object
    Triangle vec_tria_mov;
    Vector vec_TriaFaceNormal;
    vector<Vector> vec_tria_Edges;
    Vector axis_e_Edge2;
    bool temp_status;
    vector<Vector> vec_es = {Vector(1,0,0), Vector(0,1,0), Vector(0,0,1)};
    double scaling = Vector::Norm(Vector(SMOOTHING_RADIUS, SMOOTHING_RADIUS, SMOOTHING_RADIUS));

    clock_t tStart = clock();


    for(auto const &triaIdPair: mesh_Tria.GetMesh()) {

        const Triangle &tria = triaIdPair.second;


        //PRECOMPUTING TRIANGLE NORMAL AND EDGES - START (Triangle translation during SAT has no impact on the edge vector. Edge does not have to be recomputed for each Tria Box intersection)
        vec_TriaFaceNormal = tria.GetNormal(tria);

        vec_tria_Edges.push_back(tria.GetEdge(tria, 0));
        vec_tria_Edges.push_back(tria.GetEdge(tria, 1));
        vec_tria_Edges.push_back(tria.GetEdge(tria, 2));
        //PRECOMPUTING TRIANGLE NORMAL AND EDGES - END

        //Parent cell represents the cell which contains the centroid of the tria being tested for intersection.
        vector<int> parentCellId = tria.GetParentCellByCentroid(gridOrigin, G1.GetXstep(), G1.GetYstep(), G1.GetZstep(), tria);
        Bbox_parentCell = Bbox(parentCellId, G1.GetGridOrigin(), G1.GetXstep(), G1.GetYstep(), G1.GetZstep());
        std::ostringstream filename;
        filename << "./../../../../OUTPUTS/PARENT_CELL/TEST_PARENT_CELL" << parentCellId[0] << parentCellId[1] << parentCellId[2] << ".stl";
        Bbox_parentCell.Create_box_stl(filename.str(), Bbox_parentCell.GetMin(), Bbox_parentCell.GetMax());

        //Append parent cell ID with this triangle (If triangle centroid is inside the parent cell then there is definite overlap)
        G1.InsertToCellTriaMap(parentCellId, tria);
        //

        //Fetching neighbours to parent cell
        vector<vector<int>> neighbours = G1.GetNeighbours(parentCellId);

        // Broad phase collision check followed by narrow phase for neighbours of parent cell
        tria_BV = tria.BoundingVolume(tria);
        tria_BV.Create_box_stl("./../../../OUTPUTS/TEST_TRIA_BV.stl", tria_BV.GetMin(), tria_BV.GetMax());
        Bbox tmp_BboxNeighbour;

        // Single Triangle V/s all neighbours checked using bounding volume - START

        int i = 0;
        for(auto &neighbour: neighbours) {

            std::ostringstream filename_NonExtended;

            filename_NonExtended << "./../../../OUTPUTS/NONEXTENDED_BOX/TEST_NONEXTENDED_BOX_" << parentCellId[0] << " " << parentCellId[1] << " " << parentCellId[2] << "_" <<  neighbour[0] << neighbour[1] << neighbour[2] << ".stl";
            tmp_BboxNeighbour = Bbox(neighbour, G1.GetGridOrigin(), G1.GetXstep(), G1.GetYstep(), G1.GetZstep());
            tmp_BboxNeighbour.Create_box_stl(filename_NonExtended.str(), tmp_BboxNeighbour.GetMin(), tmp_BboxNeighbour.GetMax());


            std::ostringstream filename;
            filename << "./../../../OUTPUTS/EXTENDED_BOX/TEST_EXTENDED_BOX_" << neighbour[0] << neighbour[1] << neighbour[2] << ".stl";
            tmp_BboxNeighbour = tmp_BboxNeighbour.ExtendBox(tmp_BboxNeighbour, scaling);
            tmp_BboxNeighbour.Create_box_stl(filename.str(), tmp_BboxNeighbour.GetMin(), tmp_BboxNeighbour.GetMax());
            i++;

            // Can be optimized more by consdering only those cells which lie within the circumsphere of the triangle

            // Checking if bounding volume of Triangle overlaps with expanded cell. If not overlapping continue to next neighbour else perform SAT TEST
            if(tmp_BboxNeighbour.IsOverlappingBbox(tmp_BboxNeighbour, tria_BV)) {

                //Starting SAT test
                //Compute the half length vector of expanded cell. (vec_h has absolute values)
                vec_h = tmp_BboxNeighbour.ComputeHalfLength(tmp_BboxNeighbour);

                //Compute centroid(Required to move box to centre and triangle relative to box)
                vec_BoxCentre = Vector(tmp_BboxNeighbour.GetCentroid().GetX(),tmp_BboxNeighbour.GetCentroid().GetY(), tmp_BboxNeighbour.GetCentroid().GetZ());

                // Translate triangle as relative to AABB moved to origin(0,0,0) (Translation is done towards origin)
                vec_tria_mov = tria.Translate(tria, vec_BoxCentre);

                // Individual Triagle V/s Individual box - START
                // Compute test axes and check for overlap
                // 9 Text axes from edge cross product
                int k=0;
                for(auto &vec_e: vec_es) {

                    for(auto &vec_tria_Edge2 : vec_tria_Edges) {

                        // (e0 X Edge0)=0,(e0 X Edge1)=1,(e0 X Edge2) = 2, ........
                        axis_e_Edge2 = Vector::CrossProduct(vec_e,vec_tria_Edge2);

                        /*
                        Overlap test

                        Axis normalization not necessary since both
                        triangle vertices and AABB half length are
                        dotted with axis without normalization
                        */

                        temp_status = PerformOverlapTest2(vec_tria_mov,axis_e_Edge2,vec_h);

                        if(!temp_status) {

                            if(!SILENT_OVERLAP) {
                                cout << "NO OVERLAP ON CROSS PRODUCT AXIS_" << k << endl;
                            }

                            break;

                        }


                        k++;

                    }

                    if(!temp_status) {

                        break;

                    }

                }

                if(!temp_status) {

                    //No overlap of current triangle with current neighbour. Continue to next neighbour of same triangle. If neigbhour being
                    // checked is the last neighbour then go to the next triangle (Handled by foreach loop)
                    continue;

                }

                // Test axes corresponding to the face normals of AABB
                // X,Y,Z-Axis
                if(!(PerformOverlapTest2_AABBnormals(vec_tria_mov, vec_h))) {

                    if(!SILENT_OVERLAP) {
                        cout << "NO OVERLAP ON AABB Axis of cell " << endl;
                    }

                    continue;

                }

                // Test Triangle face normal (AABB Plane intersection)
                temp_status = PerformOverlapTest2(vec_tria_mov,vec_TriaFaceNormal,vec_h);

                if(!temp_status) {

                    if(!SILENT_OVERLAP) {
                        cout << "NO OVERLAP ON AABB Axis of cell " << endl;
                    }

                    continue;

                }
                if(!SILENT_OVERLAP) {
                    cout << "OVERLAP FOUND" << endl;
                }

                //Append neighbour cell ID with this triangle
                G1.InsertToCellTriaMap(neighbour, tria);
                //

                // Individual Triagle V/s Individual box - END
            }

        }

    }


    cout << "STARTING PARTICLE GENERATION" << endl;
    clock_t particle_generation_clock = clock();
    ParticleGenerator particles = ParticleGenerator(MASS_PARTICLE, SMOOTHING_RADIUS);
    particles.CreateParticlesByBbox(0.5,1,1,2.5,2.3,2,particles.particle_radius_,0.01);
    map<int,FluidParticle> particleIdMap = particles.GetParticles();
    printf("Time taken for particle generation: %.2fs\n", (double)(clock() - particle_generation_clock)/CLOCKS_PER_SEC);

    cout << "STARTING DEM CORE" <<endl;
    clock_t DEM_clock = clock();
    dem_core environment = dem_core(particleIdMap, G1.GetCellTriaMap(),G1, 10000, 1000);
    environment.StartSim(10000, 0.001);
    printf("Time taken for simulation: %.2fs\n", (double)(clock() - DEM_clock)/CLOCKS_PER_SEC);


}
