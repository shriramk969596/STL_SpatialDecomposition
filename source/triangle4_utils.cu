#include "hip/hip_runtime.h"
//
// Created by Shriram on 11/9/20.
//

#include "triangle4_utils.cuh"


using namespace CudaMathUtils;

//Gives the min and max bounding volume coordinate of a triangle. Coordinate values are held in
//a float3 array
void __device__ __host__ TriaBvMinMaxCoord(Triangle4 &tria, float3 BvMinMax[2]) {

    BvMinMax[0].x = fminf(fminf(tria.v0.x, tria.v1.x), tria.v2.x);
    BvMinMax[0].y = fminf(fminf(tria.v0.y, tria.v1.y), tria.v2.y);
    BvMinMax[0].z = fminf(fminf(tria.v0.z, tria.v1.z), tria.v2.z);

    BvMinMax[1].x = fmaxf(fmaxf(tria.v0.x, tria.v1.x), tria.v2.x);
    BvMinMax[1].y = fmaxf(fmaxf(tria.v0.y, tria.v1.y), tria.v2.y);
    BvMinMax[1].z = fmaxf(fmaxf(tria.v0.z, tria.v1.z), tria.v2.z);

}

//Gives the gird index values start and end cell of tria BV
void __device__ __host__ TriaBvMinMaxId(Triangle4 &tria, float3 gridOrigin, float3 gridSize, float3 BvMinMaxId[2]) {

    float3 BvMinMax[2];
    TriaBvMinMaxCoord(tria, BvMinMax);

    float3 min_moved = make_float3(BvMinMax[0].x - gridOrigin.x, BvMinMax[0].y - gridOrigin.y, BvMinMax[0].z - gridOrigin.z);
    float3 max_moved = make_float3(BvMinMax[1].x - gridOrigin.x, BvMinMax[1].y - gridOrigin.y, BvMinMax[1].z - gridOrigin.z);

    BvMinMaxId[0] = make_float3(int(min_moved.x/gridSize.x), int(min_moved.y/gridSize.y), int(min_moved.z/gridSize.z));

    BvMinMaxId[1] = make_float3(int(max_moved.x/gridSize.x), int(max_moved.y/gridSize.y), int(max_moved.z/gridSize.z));;

}

void __device__ __host__ TriaEdge_E0E1E2(Triangle4 &tria, float3 triaEdges[3]) {

    triaEdges[0] = make_float3(tria.v1.x - tria.v0.x, tria.v1.y - tria.v0.y, tria.v1.z - tria.v0.z);
    triaEdges[1] = make_float3(tria.v2.x - tria.v1.x, tria.v2.y - tria.v1.y, tria.v2.z - tria.v1.z);
    triaEdges[2] = make_float3(tria.v0.x - tria.v2.x, tria.v0.y - tria.v2.y, tria.v0.z - tria.v2.z);

}

bool __host__ __device__ CheckOverlap_GenericAxis(float3 translated_vertices[3], float3 &axis, const float3 &aabbHalfLength) {

    if(CudaMathUtils::Norm(axis) == 0) {

        return true;                // Check for overlap not necessary if cross product results in 0 vector

    }

    float p0 = CudaMathUtils::Dot(translated_vertices[0], axis);

    float p1 = CudaMathUtils::Dot(translated_vertices[1], axis);

    float p2 = CudaMathUtils::Dot(translated_vertices[2], axis);

    float max = fmaxf(p0, fmaxf(p1,p2));

    float min = fminf(p0, fminf(p1,p2));

    // Computing radius of AABB projected on testing axis
    float r = (aabbHalfLength.x)*abs(axis.x)+ (aabbHalfLength.y)*abs(axis.y) + (aabbHalfLength.z)*abs(axis.z);

    // Check overlap
    if(min > r || max < -r) {
        // Min and Max projected vertex of triangle lie outside projected half length of AABB
        return false;
    }

    return true;

}

bool __host__ __device__ CheckOverlap_AabbNormals(float3 translated_vertices[3], const float3 &aabbHalfLength){
    
    float max = fmaxf(translated_vertices[0].x, fmaxf(translated_vertices[1].x,translated_vertices[2].x));
    float min = fminf(translated_vertices[0].x, fminf(translated_vertices[1].x,translated_vertices[2].x));

    if(min > aabbHalfLength.x || max < -aabbHalfLength.x) {
        return false;                                              
    } 

    max = fmaxf(translated_vertices[0].y, fmaxf(translated_vertices[1].y,translated_vertices[2].y));
    min = fminf(translated_vertices[0].y, fminf(translated_vertices[1].y,translated_vertices[2].y));

    if(min > aabbHalfLength.y || max < -aabbHalfLength.y) {
        return false;                                              
    }

    max = fmaxf(translated_vertices[0].z, fmaxf(translated_vertices[1].z,translated_vertices[2].z));
    min = fminf(translated_vertices[0].z, fminf(translated_vertices[1].z,translated_vertices[2].z));

    if(min > aabbHalfLength.z || max < -aabbHalfLength.z) {
        return false;                                              
    }


    return true;

}

__global__ void  GPU_triaGridIntersection(Triangle4* tria, int* intersectedCells, float3* BvMinMaxId,
                                          float3* triaEdge_E0E1E2, float3* nxnynz, float3* gridOrigin,
                                          float3* cellSize, int* indexCounter) {

    bool temp_status = false;
//    __shared__ Triangle4 shared_tria = *tria;
    Triangle4 local_tria = *tria;
    float3 local_BvMinId = BvMinMaxId[0];
    float3 local_BvMaxId = BvMinMaxId[1];
    float3 local_triaEdge_E0E1E2[3] = {triaEdge_E0E1E2[0], triaEdge_E0E1E2[1], triaEdge_E0E1E2[2]};
    float3 local_e[3] = {make_float3(1,0,0), make_float3(0,1,0), make_float3(0,0,1)};
    float3 local_cellSize = *cellSize;
    float3 local_aabbHalfLength = make_float3(local_cellSize.x/2.0f, local_cellSize.y/2.0f, local_cellSize.z/2.0f );


    //Global Thread ID
    unsigned int tid_x =  blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int tid_y =  blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int tid_z =  blockIdx.z * blockDim.z + threadIdx.z;
    int arrIndex = CellHash(tid_x, tid_y, tid_z, GridParam::nx, GridParam::ny, GridParam::nz);
    intersectedCells[arrIndex] = -1;
    printf("%d",arrIndex);

    unsigned int cellId_x = tid_x + local_BvMinId.x;
    unsigned int cellId_y = tid_y + local_BvMinId.y;
    unsigned int cellId_z = tid_z + local_BvMinId.z;

    unsigned int stride_x = blockDim.x * gridDim.x;
    unsigned int stride_y = blockDim.y * gridDim.y;
    unsigned int stride_z = blockDim.z * gridDim.z;

    unsigned int index;

//    printf("Minmax %f, %f, %f\n", BvMinMaxId[0].x, BvMinMaxId[0].y, BvMinMaxId[0].z);
    printf("GLOBAL THREAD ID : %d, %d, %d GLOBAL CELL ID : %d, %d, %d\n",tid_x, tid_y, tid_z, cellId_x, cellId_y, cellId_z);
    for(int i = cellId_z; i <=local_BvMaxId.z; i += stride_z) {
        for(int j = cellId_y; j <= local_BvMaxId.y; j += stride_y) {
            for(int k = cellId_x; k <= local_BvMaxId.x; k += stride_x) {

//                printf("GLOBAL THREAD ID : %d, %d, %d GLOBAL CELL ID : %d, %d, %d i j k : %d %d %d\n",tid_x, tid_y, tid_z, cellId_x, cellId_y, cellId_z, k, j, i);
                float3 cellCentroid = CellIdToCoord(k, j, i, local_cellSize.x, local_cellSize.y, local_cellSize.z, gridOrigin->x, gridOrigin->y, gridOrigin->z);
//                printf("cellCentroid %f %f %f\n", cellCentroid.x, cellCentroid.y, cellCentroid.z);
                float3 translationVector = -cellCentroid;              //float3 vector used to translate triangle relative to cell centroid being moved to (0,0,0)
//                printf("%d %d %d translation vector %f %f %f\n",tid_x, tid_y, tid_z, translationVector.x, translationVector.y, translationVector.z);

                //Translate tria
                float3 vertex_translated[3];
                vertex_translated[0] = local_tria.v0 + translationVector;
                vertex_translated[1] = local_tria.v1 + translationVector;
                vertex_translated[2] = local_tria.v2 + translationVector;
//                printf("translated vertices %f %f %f, %f %f %f, %f %f %f\n",  vertex_translated[0].x, vertex_translated[0].y, vertex_translated[0].z,
//                                                                            vertex_translated[1].x, vertex_translated[1].y, vertex_translated[1].z,
//                                                                            vertex_translated[2].x, vertex_translated[2].y, vertex_translated[2].z);
//
                for(int cross_i = 0; cross_i < 3; cross_i++) {

                    for(int cross_j = 0; cross_j < 3; cross_j++) {

                        float3 axis = CudaMathUtils::Cross(local_e[cross_i], local_triaEdge_E0E1E2[cross_j]);
//                        printf("%d %d %d : axis = (e X Edge) => %f %f %f X %f %f %f = %f %f %f\n",  tid_x, tid_y, tid_z, local_e[cross_i].x, local_e[cross_i].y, local_e[cross_i].z,
//                                                                                                    local_triaEdge_E0E1E2[cross_j].x, local_triaEdge_E0E1E2[cross_j].y, local_triaEdge_E0E1E2[cross_j].z,
//                                                                                                    axis.x, axis.y, axis.z);

                        /*
                        Overlap test

                        Axis normalization not necessary since both
                        triangle vertices and AABB half length are
                        dotted with axis without normalization
                        */
                        temp_status = CheckOverlap_GenericAxis(vertex_translated, axis, local_aabbHalfLength);
//
                        if( !temp_status) {
                            break;
                        }

                    }

                    if(!temp_status) {
                        break;
                    }

                }

                if(!temp_status) {
                    continue;
                }

                // Test axes corresponding to the face normals of AABB
                // X,Y,Z-Axis
                if(!(CheckOverlap_AabbNormals(vertex_translated, local_aabbHalfLength))) {
                    continue;
                }

                // Test Triangle face normal (AABB Plane intersection)
                if(!(CheckOverlap_GenericAxis(vertex_translated, local_tria.n, local_aabbHalfLength))) {
                    continue;
                }

                //All tests for current tria passed. Add ID to cell tria map
                int cellIndex = CellHash(k, j, i, GridParam::nx, GridParam::ny, GridParam::nz);

//                index = atomicAdd(indexCounter, 1);
//                printf("INDEX COUNTER %d\n", index);
//                intersectedCells[arrIndex] = cellIndex; // change to intersectedCells[hash from thread id] = [hash from cell id];
//                printf("%d %d %d : %d\n", cellId_x, cellId_y, cellId_z, cellIndex);

            }
        }
    }

    //compute cell index from tid
}






//TODO: MAKE THIS A KERNEL AND LAUNCH A THREAD PER TRIANGLE IF n_CELLS < n_TRIAS
//checks the passed tria against all the cells for intersection in the grid.
void __host__ TriaLocalGridIntersection(Triangle4 &tria, int triaId ,std::vector<std::vector<int>> &cellToTriaMap) {
    std::cout << "NEW TRIA : " << triaId << std::endl;
    bool temp_status = false;

    float3 BvMinMaxId[2];

    //get local cell start and end ID based on triangle bounding volume
    TriaBvMinMaxId(tria, make_float3(GridParam::grid_origin_x, GridParam::grid_origin_y, GridParam::grid_origin_z), make_float3(GridParam::delX, GridParam::delY, GridParam::delZ), BvMinMaxId);

    //Launch Cell intersection on CPU or GPU based on number of cells to test(TODO: Profile code to get optimum number of cells)

    //If min and max of tri BV is within one cell append cell ID to cellToTriaMap and return. SAT not necessary.
    if(CudaMathUtils::equalTo(BvMinMaxId[0], BvMinMaxId[1])) {
        int cellIndex = CellHash(BvMinMaxId[0].x, BvMinMaxId[0].y, BvMinMaxId[0].z, GridParam::nx, GridParam::ny, GridParam::nz);
        cellToTriaMap[cellIndex].push_back(triaId);
        std::cout << "Kernel launch not required. Tria belongs to single cell : " << triaId << std::endl;
        return;
    }

    float3 triaEdges_E0E1E2[3];
    TriaEdge_E0E1E2(tria, triaEdges_E0E1E2);

    unsigned int n_cellCheckCount = (BvMinMaxId[1].x - BvMinMaxId[0].x + 1) * (BvMinMaxId[1].y - BvMinMaxId[0].y + 1) * (BvMinMaxId[1].z - BvMinMaxId[0].z + 1);

//    GPU(If number of cells to be checked exceeds threshold)
//TODO: Needs testing
    if(n_cellCheckCount >= 1) {

        float3* d_nxnynz;
        const unsigned int NUM_CELLS_BYTES = sizeof(d_nxnynz);
        CudaSafeCall(hipMallocManaged(&d_nxnynz, NUM_CELLS_BYTES));
        d_nxnynz->x = GridParam::nx; d_nxnynz->y = GridParam::ny; d_nxnynz->z = GridParam::nz;

        float3* d_gridOrigin;
        const unsigned int GRID_ORIGIN_BYTES = sizeof(d_gridOrigin);
        CudaSafeCall(hipMallocManaged(&d_gridOrigin, GRID_ORIGIN_BYTES));
        d_gridOrigin->x = GridParam::grid_origin_x; d_gridOrigin->y = GridParam::grid_origin_y; d_gridOrigin->z = GridParam::grid_origin_z;

        float3* d_cellSize;
        const unsigned int CELL_SIZE_BYTES = sizeof(d_cellSize);
        CudaSafeCall(hipMallocManaged(&d_cellSize, CELL_SIZE_BYTES));
        d_cellSize->x = GridParam::delX; d_cellSize->y = GridParam::delY; d_cellSize->z = GridParam::delZ;

        float3* d_triaEdge_E0E1E2;
        const unsigned int EDGE_ARRAY_BYTES = sizeof(triaEdges_E0E1E2);
        CudaSafeCall(hipMalloc(&d_triaEdge_E0E1E2, EDGE_ARRAY_BYTES));
        CudaSafeCall(hipMemcpy(d_triaEdge_E0E1E2, &triaEdges_E0E1E2, EDGE_ARRAY_BYTES, hipMemcpyHostToDevice));

        int* d_intersectedCells;
        const unsigned int CELL_COUNT = n_cellCheckCount;
        const unsigned int ARRAY_BYTES = CELL_COUNT * sizeof(int*);
        CudaSafeCall(hipMallocManaged(&d_intersectedCells, ARRAY_BYTES));
//        CudaSafeCall(hipMemset(d_intersectedCells, -1, ARRAY_BYTES));

        Triangle4* d_tria;
        const unsigned int TRIA_BYTES = sizeof(Triangle4);
        CudaSafeCall(hipMalloc(&d_tria, TRIA_BYTES));
        CudaSafeCall(hipMemcpy(d_tria, &tria, TRIA_BYTES, hipMemcpyHostToDevice));


        float3* d_BvMinMaxId;
        const unsigned int MINMAX_ARRAY_BYTES = sizeof(BvMinMaxId);
        CudaSafeCall(hipMallocManaged(&d_BvMinMaxId, MINMAX_ARRAY_BYTES));
        d_BvMinMaxId[0].x = BvMinMaxId[0].x; d_BvMinMaxId[0].y = BvMinMaxId[0].y; d_BvMinMaxId[0].z = BvMinMaxId[0].z;
        d_BvMinMaxId[1].x = BvMinMaxId[1].x; d_BvMinMaxId[1].y = BvMinMaxId[1].y; d_BvMinMaxId[1].z = BvMinMaxId[1].z;

        unsigned int n_threads = n_cellCheckCount;

        int* d_indexCounter;
        CudaSafeCall(hipMallocManaged(&d_indexCounter, sizeof(int)));
        *d_indexCounter = -1;

        int n_threads_x = BvMinMaxId[1].x - BvMinMaxId[0].x + 1;
        int n_threads_y = BvMinMaxId[1].y - BvMinMaxId[0].y + 1;
        int n_threads_z = BvMinMaxId[1].z - BvMinMaxId[0].z + 1;

        dim3 threadBlock(8,8,8); //make round multiple of warp size wiz 32.
        dim3 grid((n_threads_x+threadBlock.x-1)/threadBlock.x, (n_threads_y+threadBlock.y-1)/threadBlock.y, (n_threads_z+threadBlock.z-1)/threadBlock.z);

//        int blockSize;      // The launch configurator returned block size
//        int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch
//        int gridSize;       // The actual grid size needed, based on input size
//        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, GPU_triaGridIntersection, 0, n_cellCheckCount);
//        gridSize = (n_cellCheckCount + blockSize - 1) / blockSize;

        GPU_triaGridIntersection<<<dim3(1,1,1), dim3(1,1,1)>>>(d_tria, d_intersectedCells, d_BvMinMaxId, d_triaEdge_E0E1E2, d_nxnynz, d_gridOrigin, d_cellSize, d_indexCounter);
        hipDeviceSynchronize();

        printf("%d %d", d_intersectedCells[0], d_intersectedCells[1]);

        for(int i = 0; i < CELL_COUNT; i++) {
            if(d_intersectedCells[i] == -1) {
                continue;
            }
            cellToTriaMap[d_intersectedCells[i]].push_back(triaId);
        }

        CudaSafeCall(hipFree(d_tria));
        CudaSafeCall(hipFree(d_intersectedCells));
        CudaSafeCall(hipFree(d_BvMinMaxId));
        CudaSafeCall(hipFree(d_triaEdge_E0E1E2));
        CudaSafeCall(hipFree(d_nxnynz));
        CudaSafeCall(hipFree(d_gridOrigin));
        CudaSafeCall(hipFree(d_cellSize));
        CudaSafeCall(hipFree(d_indexCounter));
    }

    //CPU tria intersection
    for(int i = BvMinMaxId[0].z; i <= BvMinMaxId[1].z; i++) {
        for(int j = BvMinMaxId[0].y; j <= BvMinMaxId[1].y; j++) {
            for(int k = BvMinMaxId[0].x; k <= BvMinMaxId[1].x; k++) {

                float3 cellCentroid = CellIdToCoord(k, j, i, GridParam::delX, GridParam::delY, GridParam::delZ, GridParam::grid_origin_x, GridParam::grid_origin_y, GridParam::grid_origin_z);
                float3 translationVector = -cellCentroid;              //float3 vector used to translate triangle relative to cell centroid being moved to (0,0,0)

                //Translate tria
                float3 vertex_translated[3];
                vertex_translated[0] = tria.v0 + translationVector;
                vertex_translated[1] = tria.v1 + translationVector;
                vertex_translated[2] = tria.v2 + translationVector;

                
                for(int cross_i = 0; cross_i < 3; cross_i++) {

                    for(int cross_j = 0; cross_j < 3; cross_j++) {

                        float3 axis = CudaMathUtils::Cross(SatParam::e[cross_i], triaEdges_E0E1E2[cross_j]);

                        /*
                        Overlap test

                        Axis normalization not necessary since both
                        triangle vertices and AABB half length are
                        dotted with axis without normalization
                        */
                        temp_status = CheckOverlap_GenericAxis(vertex_translated, axis, SatParam::aabbHalfLength);

                        if( !temp_status) {
                            break;
                        }

                    }

                    if(!temp_status) {
                        break;
                    }

                }

                if(!temp_status) {
                    continue;
                }

                // Test axes corresponding to the face normals of AABB
                // X,Y,Z-Axis
                if(!(CheckOverlap_AabbNormals(vertex_translated, SatParam::aabbHalfLength))) {
                    continue;
                }

                // Test Triangle face normal (AABB Plane intersection)
                if(!(CheckOverlap_GenericAxis(vertex_translated, tria.n, SatParam::aabbHalfLength))) {
                    continue;
                }

                //All tests for current tria passed. Add ID to cell tria map
                int cellIndex = CellHash(k, j, i, GridParam::nx, GridParam::ny, GridParam::nz);
                std::cout << k << " " << j << " " << i << " : " << cellIndex << std::endl;
//                cellToTriaMap[cellIndex].push_back(triaId);
            }
        }

    }

    return;
}

//Generates spatial decomposition data for input stl mesh in a form viable for GPU usage.
void __host__ GenerateMeshSpatialDecompositionData(std::vector<std::vector<int>> &cellToTriaMap,
                                                   std::vector<unsigned int> &view_triaIds_flat,
                                                   std::vector<unsigned int>   &view_numTriasPerCell,
                                                   std::vector<unsigned int>  &view_cellStart) {

    unsigned int start = 0;
    unsigned int tria_count = 0;
    unsigned int count = 0;
    for(auto cell : cellToTriaMap) {

        unsigned int offset = 0;
        for(unsigned int triaId: cell) {

//            triaIds_flat.push_back(triaId);
            view_triaIds_flat[tria_count] = triaId;

            tria_count++;
            offset++;
        }
//        numTriasPerCell.push_back(offset);
        view_numTriasPerCell[count] = offset;

//        cellStart.push_back(start);
        view_cellStart[count] = start;
        count++;
        start = start + offset;
    }

}